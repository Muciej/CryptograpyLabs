#include <iostream>
#include <MD5Modified/md5Attack.cuh>
#include <MD5Modified/md5Utils.hpp>
#include <MD5Modified/data.hpp>

void fullMain()
{
	uint32_t* msg0;
	uint32_t* msg0Prime;
	uint32_t* msg1;
	uint32_t* msg1Prime;

    hipMallocManaged(&msg0, 16 * sizeof(uint32_t));
    hipMallocManaged(&msg0Prime, 16 * sizeof(uint32_t));
    hipMallocManaged(&msg1, 16 * sizeof(uint32_t));
    hipMallocManaged(&msg1Prime, 16 * sizeof(uint32_t));

    MD5Modified::fullAttack(msg0, msg0Prime, msg1, msg1Prime);

    MD5Modified::utils::printMessage(msg0);
    MD5Modified::utils::printMessage(msg0Prime);
    MD5Modified::utils::printMessage(msg1);
    MD5Modified::utils::printMessage(msg1Prime);
}

void onlyEx3()
{
	uint32_t* msg0;
	uint32_t* msg0Prime;
	uint32_t* msg1;
	uint32_t* msg1Prime;

    hipMallocManaged(&msg0, 16 * sizeof(uint32_t));
    hipMallocManaged(&msg0Prime, 16 * sizeof(uint32_t));
    hipMallocManaged(&msg1, 16 * sizeof(uint32_t));
    hipMallocManaged(&msg1Prime, 16 * sizeof(uint32_t));

	for (uint16_t i = 0; i < 16; i++) {
		msg0[i] = FirstPair::msg_0[i];
		msg0Prime[i] = FirstPair::msg_0_prime[i];
	}

    MD5Modified::onlySecondPart(msg0, msg0Prime, msg1, msg1Prime);

    MD5Modified::utils::printMessage(msg1);
    MD5Modified::utils::printMessage(msg1Prime);
}

int main()
{
    // fullMain();
    onlyEx3();

    return 0;
}