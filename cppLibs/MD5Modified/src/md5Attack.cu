#include "hip/hip_runtime.h"
#include <cstdint>
#include <hiprand/hiprand_kernel.h>

#include <MD5Modified/deviceConstants.cuh>
#include <MD5Modified/md5MessageModification.cuh>

using std::uint32_t;

namespace MD5Modified
{
    using namespace Constants;

    __device__ void getWordDifference(const WordTable a, const WordTable b, WordTable& difference)
    {
        for(int i = 0; i < 4; i++)
        {
            difference[i] = a[i] - b[i];
        }
    }

    __device__ void getMessageDifference(const uint32_t* msgA, const uint32_t* msgB, uint32_t* difference)
    {
        for(int i = 0; i < 16; i++)
        {
            difference[i] = msgA[i] - msgB[i];
        }
    }

    __device__ bool checkDesiredDifference(const uint32_t* msgA, const uint32_t* msgB, const uint32_t* desiredDifference)
    {
        bool isGood = true;
        for(int i = 0; i < 16; i++)
        {
            if(desiredDifference[i] != msgA[i] - msgB[i])
            {
                isGood = false;
            }
        }
        return isGood;
    }

    __device__ std::uint32_t func(int it, std::uint32_t abcd[4])
    {
        switch (it)
        {
        case 0:
            return (abcd[1] & abcd[2]) | (~abcd[1] & abcd[3]);
            break;
        case 1:
            return (abcd[3] & abcd[1]) | (~abcd[3] & abcd[2]);
            break;
        case 2:
            return  abcd[1] ^ abcd[2] ^ abcd[3];
            break;
        case 3:
            return abcd[2] ^ (abcd[1] | ~abcd[3]);
            break;
        }
        return 0;
    }

    __device__ std::uint32_t rol(std::uint32_t r, std::uint8_t N)
    {
        return (r << N) | (r >> (32 - N));
    }

    __device__ std::uint32_t unrol(std::uint32_t r, std::uint8_t N)
    {
        return (r >> N) | (r << (32 - N));
    }

    __device__ void md5(const WordTable h0, const std::uint32_t* msg, WordTable h)
    {
        WordTable state;
        std::int16_t m, o, inputInd;
        std::uint32_t f;
        const std::uint8_t* rotn;

        for (int i = 0; i < 4; i++)
        {
            state[i] = h[i] = h0[i];			// init with start vector
        }
        for (int mainIteration = 0; mainIteration < 4; mainIteration++)
        {
            rotn = rots[mainIteration];
            m = M[mainIteration]; o = O[mainIteration];
            for (int q = 0; q < 16; q++)
            {
                inputInd = (m * q + o) % 16;
                f = state[1] + rol(state[0] + func(mainIteration, state) + constTable[q + 16 * mainIteration] + msg[inputInd], rotn[q % 4]);

                state[0] = state[3];
                state[3] = state[2];
                state[2] = state[1];
                state[1] = f;
            }
        }
        for (int i = 0; i<4; i++)
        {
            h[i] += state[i];				// init state + state after iterations
        }
    }

    __device__ void md5FirstPart(uint32_t* msg, WordTable h)
    {
        WordTable state;
        std::int16_t m, o, inputInd;
        std::uint32_t f;
        const std::uint8_t* rotn;

        for (int i = 0; i < 4; i++)
        {
            state[i] = h[i] = def_h0[i];			// init with default start vector
        }

        rotn = rots[0];
        m = M[0]; o = O[0];
        for (int q = 0; q < 16; q++)
        {
            inputInd = (m * q + o) % 16;
            f = state[1] + rol(state[0] + func(0, state) + constTable[q] + msg[inputInd], rotn[q % 4]);

            // MODIFICATION
            FirstPart::modifications[q](f, state);
            msg[inputInd] = unrol(f - state[1], rotn[q % 4]);

            state[0] = state[3];
            state[3] = state[2];
            state[2] = state[1];
            state[1] = f;
        }

        for (int mainIteration = 1; mainIteration < 4; mainIteration++)
        {
            rotn = rots[mainIteration];
            m = M[mainIteration]; o = O[mainIteration];
            for (int q = 0; q < 16; q++)
            {
                inputInd = (m * q + o) % 16;
                f = state[1] + rol(state[0] + func(mainIteration, state) + constTable[q + 16 * mainIteration] + msg[inputInd], rotn[q % 4]);

                state[0] = state[3];
                state[3] = state[2];
                state[2] = state[1];
                state[1] = f;
            }
        }
        for (int i = 0; i<4; i++)
        {
            h[i] += state[i];				// init state + state after iterations
        }
    }

    __device__ void md5SecondPart(const WordTable h0, std::uint32_t* msg, uint32_t* h)
    {
        uint32_t* state;
        std::int16_t m, o, inputInd;
        std::uint32_t f;
        const std::uint8_t* rotn;

        for (int i = 0; i < 4; i++)
        {
            state[i] = h[i] = h0[i];			// init with default start vector
        }

        rotn = rots[0];
        m = M[0]; o = O[0];
        for (int q = 0; q < 16; q++)
        {
            inputInd = (m * q + o) % 16;
            f = state[1] + rol(state[0] + func(0, state) + constTable[q] + msg[inputInd], rotn[q % 4]);

            // MODIFICATION
            SecondPart::modifications[q](f, state);
            msg[inputInd] = unrol(f - state[1], rotn[q % 4]);

            state[0] = state[3];
            state[3] = state[2];
            state[2] = state[1];
            state[1] = f;
        }

        for (int mainIteration = 1; mainIteration < 4; mainIteration++)
        {
            rotn = rots[mainIteration];
            m = M[mainIteration]; o = O[mainIteration];
            for (int q = 0; q < 16; q++)
            {
                inputInd = (m * q + o) % 16;
                f = state[1] + rol(state[0] + func(mainIteration, state) + constTable[q + 16 * mainIteration] + msg[inputInd], rotn[q % 4]);

                state[0] = state[3];
                state[3] = state[2];
                state[2] = state[1];
                state[1] = f;
            }
        }
        for (int i = 0; i<4; i++)
        {
            h[i] += state[i];				// init state + state after iterations
        }
    }

    __global__ void attackFirstPart(uint32_t* msg0, uint32_t* msg0Prime, uint32_t* result, uint32_t* resultPrime, int* isDifferentialFound)
    {
        // initialize CUDA random generator
        const uint32_t idx = threadIdx.x + blockIdx.x * blockDim.x;

        hiprandState generator{};
        hiprand_init(idx, idx, 0, &generator);

        // init variables
        uint32_t msg0Local[16];
        uint32_t msg0PrimeLocal[16];

        // calculate MD5 with message modification for round one
        while(*isDifferentialFound <= 0)
        {
            // random message selection
            for(auto& word : msg0Local)
            {
                word = hiprand(&generator);
            }

            md5FirstPart(msg0Local, result);

            for(int i = 0; i < 16; i++)
            {
                msg0PrimeLocal[i] = msg0Local[i] + Constants::deltaM0[i];
            }

            if(checkDesiredDifference(resultPrime, result, Constants::deltaH1))
            {
                atomicAdd(isDifferentialFound, 1);
            }
        }

        for(int i = 0; i < 16; i++)
        {
            msg0[i] = msg0Local[i];
            msg0Prime[i] = msg0PrimeLocal[i];
        }
    }

    __global__ void attackSecondPart(uint32_t* msg1, uint32_t* msg1Prime, const uint32_t* IV, const uint32_t* IVPrime, int* isCollisionFound)
    {
        // initialize CUDA random generator
        const uint32_t idx = threadIdx.x + blockIdx.x * blockDim.x;

        hiprandState generator{};
        hiprand_init(idx, idx, 0, &generator);

        // init variables
        uint32_t msg1Local[16];
        uint32_t msg1PrimeLocal[16];
        uint32_t result[4];
        uint32_t resultPrime[4];

        // calculate MD5 with message modification for round one
        while(*isCollisionFound <= 0)
        {
            // random message selection
            for(auto& word : msg1Local)
            {
                word = hiprand(&generator);
            }

            md5SecondPart(IV, msg1Local, result);

            for(int i = 0; i < 16; i++)
            {
                msg1PrimeLocal[i] = msg1Local[i] + Constants::deltaM1[i];
            }

            md5(IVPrime, msg1PrimeLocal, resultPrime);

            bool areEqual = true;
            for (int i = 0; i < 4; i++) {
                if (result[i] != resultPrime[i]) {
                    areEqual = false;
                }
            }

            if(areEqual)
            {
                atomicAdd(isCollisionFound, 1);
            }
        }

        for(int i = 0; i < 16; i++)
        {
            msg1[i] = msg1Local[i];
            msg1Prime[i] = msg1PrimeLocal[i];
        }
    }

    void fullAttack(uint32_t* msg0, uint32_t* msg0Prime, uint32_t* msg1, uint32_t* msg1Prime)
    {
        uint32_t* result;
        uint32_t* resultPrime;
        int* isCollisionFound;
        int* isDifferentialFound;

        // allocate CUDA memory
        hipMallocManaged(&result, 4 * sizeof(uint32_t));
        hipMallocManaged(&resultPrime, 4 * sizeof(uint32_t));
        hipMallocManaged(&isCollisionFound, sizeof(int));
        hipMallocManaged(&isDifferentialFound, sizeof(int));

        // run attack in a loop
        while(*isCollisionFound <= 0)
        {
            *isDifferentialFound = 0;
            attackFirstPart<<<32, 256>>>(msg0, msg0Prime, msg1, msg1Prime, isDifferentialFound);
            attackSecondPart<<<32, 256>>>(msg1, msg1Prime, result, resultPrime, isCollisionFound);
        }
    }

} // namespace MD5Modified